#include "hip/hip_runtime.h"
#include "solve.h"
#include <iostream>
#include <hip/hip_runtime.h>

#define CHECK_CUDA_CALL(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << " at " << __LINE__ << std::endl; \
        exit(1); \
    } \
}

__global__ void vector_add(const float* A, const float* B, float* C, int N) {
    int idx = blockIdx.x + blockDim.x + threadIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
        printf("Thread %d: A=%f, B=%f, C=%f\n", idx, A[idx], B[idx], C[idx]);
    }
}

void solve(const float* A, const float* B, float* C, int N) {
    float *d_A, *d_B, *d_C;

    // Allocate device memory
    CHECK_CUDA_CALL(hipMalloc(&d_A, N * sizeof(float)));
    CHECK_CUDA_CALL(hipMalloc(&d_B, N * sizeof(float)));
    CHECK_CUDA_CALL(hipMalloc(&d_C, N * sizeof(float)));

    // Copy input data from host to device
    CHECK_CUDA_CALL(hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA_CALL(hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice));

    // Calculate grid and block dimensions
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    vector_add<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    CHECK_CUDA_CALL(hipDeviceSynchronize());
    CHECK_CUDA_CALL(hipGetLastError());

    // Copy result back to host
    CHECK_CUDA_CALL(hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost));

    // Free device memory
    CHECK_CUDA_CALL(hipFree(d_A));
    CHECK_CUDA_CALL(hipFree(d_B));
    CHECK_CUDA_CALL(hipFree(d_C));
}
