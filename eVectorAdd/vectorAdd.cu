#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>


__global__ void vector_add(const float* A, const float* B, float* C, int N) {

}

void solve(const float* A, const float* B, float* C, int N) {
    float *d_A, *d_B, *d_C;

    // Allocate device memory
    hipMalloc(&d_A, N * sizeof(float));
    hipMalloc(&d_B, N * sizeof(float));
    hipMalloc(&d_C, N * sizeof(float));

    // Copy input data from host to device
    hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice);

    // Calculate grid and block dimensions
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    vector_add<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipGetLastError();
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost));

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

void main() {


}
