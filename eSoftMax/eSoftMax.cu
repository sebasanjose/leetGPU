#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void softmax_kernel(const float* input, float* output, int N) {
    // TODO: Implement the softmax kernel.  Remember to use the max trick!
}

void solve(const float* input, float* output, int N) {
    float *d_input, *d_output;

    // Allocate device memory
    hipMalloc(&d_input, N * sizeof(float));
    hipMalloc(&d_output, N * sizeof(float));

    // Copy input data from host to device
    hipMemcpy(d_input, input, N * sizeof(float), hipMemcpyHostToDevice);

    // Calculate grid and block dimensions
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    softmax_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, N);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(output, d_output, N * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
}