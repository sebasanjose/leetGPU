#include "hip/hip_runtime.h"
#include "solve.h"
#include <hip/hip_runtime.h>

__global__ void matrix_transpose(const float* input, float* output, int rows, int cols) {

}

void solve(const float* input, float* output, int rows, int cols) {
    float *d_input, *d_output;

    // Allocate device memory
    hipMalloc(&d_input, rows * cols * sizeof(float));
    hipMalloc(&d_output, rows * cols * sizeof(float));

    // Copy input data from host to device
    hipMemcpy(d_input, input, rows * cols * sizeof(float), hipMemcpyHostToDevice);

    // Calculate grid and block dimensions
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid(
        (cols + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (rows + threadsPerBlock.y - 1) / threadsPerBlock.y
    );

    // Launch the kernel
    matrix_transpose<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, rows, cols);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(output, d_output, rows * cols * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_input);
    hipFree(d_output);
}